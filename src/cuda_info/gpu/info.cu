#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void CudaDeviceInfo() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);

};

extern "C" void hello(){
  printf("hello");
}

int main(){
  CudaDeviceInfo();
  hipDeviceSynchronize();
}