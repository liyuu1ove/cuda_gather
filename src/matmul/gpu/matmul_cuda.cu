#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h> // 确保包含这个头文件
#include <math.h>
#include "error_check.h"

const int block_M=16;
const int block_N=16;
const int block_K=16;
__global__ void matmul_kernel_fp32(
    float* device_A,
    float* device_B,
    float* device_C,
    int M, int K, int N){//A: M*K B: K*N C: M*N
        int col = blockIdx.x*blockDim.x +threadIdx.x;
        int row = blockIdx.y*blockDim.y + threadIdx.y;
        float tmp=0;
        if(col<N&&row<M){
            for(int i=0;i<K;i++){
                tmp+=device_A[K*row+i]*device_B[K*col+i];
            }
            device_C[col*N+M]=tmp;
        }
    }

extern "C" void matmul_cuda_fp32(void* const host_A,
    void* const host_B,void* const host_C,const int M,const int K,const int N){
        int num_block_x = (M) / (block_M)+1;
        int num_block_y = (N) / (block_N)+1;
        dim3 grid_dim(num_block_x,num_block_y);
        dim3 block_dim(block_M,block_N);
        matmul_kernel_fp32<<<grid_dim, block_dim>>>((float*) host_A,
    (float*) host_B,(float*) host_C,M,K,N);
    }